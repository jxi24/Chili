#include "Cuda/Interface.cuh"


namespace apes {

// hipError_t CudaCheckErrorBase(const hipError_t err, const char *file, int line) {
// }

hipError_t CudaCheckErrorBase(const char *file, int line) {
    return CudaCheckErrorBase(hipGetLastError(), file, line);
}

// void CudaAssertErrorBase(const hipError_t err, const char *file, int line) {
//     assert(CudaCheckErrorBase(err, file, line) == hipSuccess);
// }

void CudaAssertErrorBase(const char *file, int line) {
    CudaAssertErrorBase(hipGetLastError(), file, line);
}

hipError_t CudaMalloc(void **ptr, size_t size) {
    return hipMalloc(ptr, size);
}

hipError_t CudaMalloc(void **ptr, unsigned int size) {
    return hipMalloc(ptr, size);
}

hipError_t CudaCopyToDevice(void *target, void const *source, size_t size) {
    return hipMemcpy(target, source, size, hipMemcpyHostToDevice);
}

hipError_t CudaCopyFromDevice(void *target, void const *source, size_t size) {
    return hipMemcpy(target, source, size, hipMemcpyDeviceToHost);
}

hipError_t CudaFree(void *ptr) { return hipFree(ptr); }

}
